#include "hip/hip_runtime.h"
/** 
* André Luiz Abdalla Silveira 8030353
* Mauricio Luiz Cardoso 0000000
* 
* Esse programa escrito em CUDA visa criar um algoritmo que gera uma redução
*	de matrizes. Cada matriz é representada por um vetor e todos estão 
*	reunidos num vetor de vetores. A ideia é fazer uma função que faz uma 
*	comparação entre vetores fazendo o mínimo de operações
*
*	Observação pro Mauricio (delete depois de ler): depois de passar um tempão
*	pensando numa solução mais elegante, eu percebi que o número de comparações
*	é o mesmo se compararmos com uma implemetação onde se contrapõem dois vetores
*	iniciais, e o resultante da comparação com o próximo, e por aí vai.
*	O que falta? 1- leitura de arquivos 2- fazer a parte de CUDA 3- corrigir eventuais
*									bugs
*/

#include <stdio.h>
#define E 9 // qtde de elementos de cada matriz
/*#define linhaTam 3 */

/* pedaço de código copaido da apresentação, deve ser alterado sob demanda */
__global__ void os_menores(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void leitura (int *mat) {
	int i, j;
	for (i = 0; i < 3; i ++) {
		printf("\n");
		for (k = 0; k < 3; k ++) 
			printf("%d\t", mat[3 * i + k]);
		printf("\n");
	}
}

int * compara (int *a, int *b) { /* Muitas coisas copiadas dos slides 36 e 37 */
	int *resp, tam = E * sizeof(int);
	int *d_a, *d_b, *d_resp;
	
	// Alloc space for device copies of a, b, c
	hipMalloc((void **) &d_a, tam);
	hipMalloc((void **) &d_b, tam);
	hipMalloc((void **) &d_resp, tam);

	// Copy inputs to device
	hipMemcpy(d_a, a, tam, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, tam, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU with N threads -- mudar o valor de N
	add<<<1,N>>>(d_a, d_b, d_resp);

	// Copy result back to host
	hipMemcpy(resp, d_resp, tam, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_resp);
	return resp;
}

int * get_min (int **mat, int ini, int fim) {
	int *parteA, *parteB;
	if (ini - fim > 1){ /* mais que 2 elementos*/
		meio = (ini + fim)/2;
		parteA = get_min(mat, ini, meio);
		parteB = get_min(mat, meio + 1, fim);
		return compara(parteA, parteB);
	}else if (ini - fim == 1){ /* 2 elementos */
		return compara(mat[ini], mat[fim]);
	}
	else {
		return mat[ini];
	}
}

void apaga (int **mat){
	int i;
	int linhas = (sizeof(mat)/sizeof(int *));
	for (i = 0; i < linhas; ++i) {
		free(mat[i]);
	}
	free(mat);
}

int** alocaMatrizes(int numMatrizes) {
	int** mat = (int *) malloc(numMatrizes * sizeof(int *));
	
	for (int i = 0; i < numMatrizes; i ++){
		mat[i] = (int) malloc(E * sizeof(int));
	}
	return mat;
}

double** alocaMatrizesArquivo(FILE *arq){
    int numMatrizes, l, c;
    char* asteriscos;
    double valor;
    double **matrizes;

    fscanf(arq, "%d", &numMatrizes);
    matrizes = alocaMatrizes(numMatrizes);
    
    for(int i = 0; i < numMatrizes; i++) {
    	fscanf(arq, "%s", &asteriscos); //pula a linha de asteriscos
    	for(int j = 0; j < E; j++)
        	fscanf(arq, "%d", &matrizes[i][j]);
    }
    return matriz;
}


int main (int argc, char* argv[]) {
	if(argc != 2) {
		printf("Argumento do programa: nome do arquivo\n");
	}
	else {
		int qtde;
		int **mat;
		FILE *entrada;
		int i; # contador

		entrada =  fopen(argv[1], "r");
		if (entrada == NULL) {
			printf("Deu ruim pra abrir o arquivo");
			return EXIT_FAILURE;
		}


		fclose(entrada);
		leitura(get_min(mat, 0, qtde));
		apaga(mat);
	}
}
