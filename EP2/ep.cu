/** 
* André Luiz Abdalla Silveira 8030353
* Mauricio Luiz Cardoso 6796479
* 
* Esse programa escrito em CUDA visa criar um algoritmo que gera uma redução
*   de matrizes. Cada matriz é representada por um vetor e todos estão 
*   reunidos num vetor de vetores. A ideia é fazer uma função que faz uma 
*   comparação entre vetores fazendo o mínimo de operações
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define E 9 // qtde de elementos de cada matriz
#define linhaElementos 3 // quantidade de elementos da linha 
int numMatrizes;


__global__ void os_menores(int *d_matrizes, int posLimite, int jump) {
  int indexIni = threadIdx.x + blockIdx.x * blockDim.x;

  for(int i = indexIni; i < posLimite; i += jump)
    if(d_matrizes[indexIni] > d_matrizes[i])
      d_matrizes[indexIni] = d_matrizes[i];

}

/* Imprime todas as matrizes de dimensão ExE contidas em matrizes*/
void leitura (int *matrizes, int numMats) {
  int i, k;
  for (i = 0; i < numMats * linhaElementos; i++) {
    for (k = 0; k < linhaElementos; k++) 
      printf("%d\t", *(matrizes++));
    printf("\n");
    if((i+1) % linhaElementos == 0)
      printf("********************\n");
  }
}


void menorMatriz(int *d_matrizes, int numMats) {
	if(numMats > 1) {
	  int numBlocks = 0;
	  int numMatResto;
    int jump = 0;
	  int numThreads = 0;
    int posLimite;

	
    // carga de tamanho de um bloco
	  if(numMats <= E * 10) {
      numMatResto = 1;
      numThreads = E;
      numBlocks = 1;
    }
    else {
      const int numMatThreads = 3; // 3 foi escolhido para que numthreads seja maior multiplo de E(tamanho de cada matriz) e menor que um warp(32)
      numThreads = E * numMatThreads; 
      int espacoTrabThre = 10 * numThreads; //cada thread devera comparar ate E * 10 matrizes
      numBlocks = E * numMats / espacoTrabThre;
      numMatResto = numBlocks * numMatThreads;
    }

    posLimite = numMats * E;
    jump = numBlocks * numThreads;
	  os_menores<<<numBlocks, numThreads>>>(d_matrizes, posLimite, jump);
    hipDeviceSynchronize();

    menorMatriz(d_matrizes, numMatResto);
	}
}


void encontraMenorMatriz(int* matrizes) {
  int tam = numMatrizes * E * sizeof(int);
  int *d_matrizes;

  // Alloc space for device copies of a, b, c
  hipMalloc((void **) &d_matrizes, tam);

  // Copy inputs to device
  hipMemcpy(d_matrizes, matrizes, tam, hipMemcpyHostToDevice);

  // encontra menor matriz
  menorMatriz(d_matrizes, numMatrizes);
  
  // Copy result back to host
  hipMemcpy(matrizes, d_matrizes, tam, hipMemcpyDeviceToHost);

  hipFree(d_matrizes);  
}


/* Le o arquivo arq que contem matrizes no formato declarado
no enunciado e retorna um vetor com todas matrizes lidas*/
int* alocaMatrizesArquivo(FILE *arq){
  char asteriscos[10];
  int *matrizes, *matrizesAux;
  fscanf(arq, "%d", &numMatrizes);
  matrizes =  (int *) malloc(E * numMatrizes * sizeof(int));

  matrizesAux = matrizes;
  for(int i = 0; i < numMatrizes; i++) {
    fscanf(arq, "%s", asteriscos); //pula a linha de asteriscos
    for(int j = 0; j < E; j++)
      fscanf(arq, "%d", matrizesAux++);
  }
  return matrizes;
}


int main (int argc, char* argv[]) {
  if(argc != 2) {
    printf("Argumento do programa: nome do arquivo\n");
  }
  else {  
    FILE *entrada;
    entrada =  fopen(argv[1], "r");
    if (entrada == NULL) {
      printf("Deu ruim pra abrir o arquivo\n");
      return EXIT_FAILURE;
    }
    int *matrizes = alocaMatrizesArquivo(entrada);
    fclose(entrada);

    encontraMenorMatriz(matrizes);
    leitura(matrizes, 1);



    // leitura(get_min(mat, 0, qtde));
    free(matrizes);
    return EXIT_SUCCESS;
  }
}

