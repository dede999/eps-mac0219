/** 
* André Luiz Abdalla Silveira 8030353
* Mauricio Luiz Cardoso 0000000
* 
* Esse programa escrito em CUDA visa criar um algoritmo que gera uma redução
*   de matrizes. Cada matriz é representada por um vetor e todos estão 
*   reunidos num vetor de vetores. A ideia é fazer uma função que faz uma 
*   comparação entre vetores fazendo o mínimo de operações
*
*   Observação pro Mauricio (delete depois de ler): depois de passar um tempão
*   pensando numa solução mais elegante, eu percebi que o número de comparações
*   é o mesmo se compararmos com uma implemetação onde se contrapõem dois vetores
*   iniciais, e o resultante da comparação com o próximo, e por aí vai.
*   O que falta? 1- leitura de arquivos 2- fazer a parte de CUDA 3- corrigir eventuais
*                                   bugs
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define E 9 // qtde de elementos de cada matriz
#define linhaElementos 3 // quantidade de elementos da linha 
int numMatrizes;

/* pedaço de código copaido da apresentação, deve ser alterado sob demanda */
__global__ void os_menores(int *a, int *b, int *c) {
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

/* Imprime todas as matrizes de dimensão ExE contidas en nat*/
void leitura (int *mat) {
    int i, k;
    for (i = 0; i < numMatrizes * linhaElementos; i++) {
        for (k = 0; k < linhaElementos; k++) 
            printf("%d\t", *(mat++));
        printf("\n");
        if((i+1) % linhaElementos == 0)
            printf("********************\n");
    }
}

int * compara (int *a, int *b) { /* Muitas coisas copiadas dos slides 36 e 37 */
    int *resp, tam = E * sizeof(int);
    int *d_a, *d_b, *d_resp;
    int N = 32;
    
    // Alloc space for device copies of a, b, c
    hipMalloc((void **) &d_a, tam);
    hipMalloc((void **) &d_b, tam);
    hipMalloc((void **) &d_resp, tam);

    // Copy inputs to device
    hipMemcpy(d_a, a, tam, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, tam, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU with N threads -- mudar o valor de N
    os_menores<<<1,N>>>(d_a, d_b, d_resp);

    // Copy result back to host
    hipMemcpy(resp, d_resp, tam, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_resp);
    return resp;
}

// int * get_min (int **mat, int ini, int fim) {
//  int *parteA, *parteB;
//  if (ini - fim > 1){ /* mais que 2 elementos*/
//      meio = (ini + fim)/2;
//      parteA = get_min(mat, ini, meio);
//      parteB = get_min(mat, meio + 1, fim);
//      return compara(parteA, parteB);
//  }else if (ini - fim == 1){ /* 2 elementos */
//      return compara(mat[ini], mat[fim]);
//  }
//  else {
//      return mat[ini];
//  }
// }

// void apaga (int **mat){
//  int i;
//  int linhas = (sizeof(mat)/sizeof(int *));
//  for (i = 0; i < linhas; ++i) {
//      free(mat[i]);
//  }
//  free(mat);
// }


/* Le o arquivo arq que contem matrizes no formato declarado
no enunciado e retorna um vetor com todas matrizes lidas*/
int* alocaMatrizesArquivo(FILE *arq){
    char asteriscos[10];
    int *matrizes, *matrizesAux;
    fscanf(arq, "%d", &numMatrizes);
    matrizes =  (int *) malloc(E * numMatrizes * sizeof(int));
    
    matrizesAux = matrizes;
    for(int i = 0; i < numMatrizes; i++) {
        fscanf(arq, "%s", asteriscos); //pula a linha de asteriscos
        for(int j = 0; j < E; j++)
            fscanf(arq, "%d", matrizesAux++);
    }
    leitura(matrizes);
    return matrizes;
}


int main (int argc, char* argv[]) {
    if(argc != 2) {
        printf("Argumento do programa: nome do arquivo\n");
    }
    else {  
        FILE *entrada;
        entrada =  fopen(argv[1], "r");
        if (entrada == NULL) {
            printf("Deu ruim pra abrir o arquivo\n");
            return EXIT_FAILURE;
        }
        int *matrizes = alocaMatrizesArquivo(entrada);

        fclose(entrada);
        // leitura(get_min(mat, 0, qtde));
        free(matrizes);
    }
}

