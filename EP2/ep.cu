#include "hip/hip_runtime.h"
/** 
* André Luiz Abdalla Silveira 8030353
* Mauricio Luiz Cardoso 6796479
* 
* Esse programa escrito em CUDA visa criar um algoritmo que gera uma redução
*   de matrizes. Cada matriz é representada por um vetor e todos estão 
*   reunidos num vetor de vetores. A ideia é fazer uma função que faz uma 
*   comparação entre vetores fazendo o mínimo de operações
*
*   Observação pro Mauricio (delete depois de ler): depois de passar um tempão
*   pensando numa solução mais elegante, eu percebi que o número de comparações
*   é o mesmo se compararmos com uma implemetação onde se contrapõem dois vetores
*   iniciais, e o resultante da comparação com o próximo, e por aí vai.
*   O que falta? 1- leitura de arquivos 2- fazer a parte de CUDA 3- corrigir eventuais
*                                   bugs
*/

#include <stdio.h>
#include <stdlib.h>
#define E 9 // qtde de elementos de cada matriz
#define linhaElementos 3 // quantidade de elementos da linha 
int numMatrizes;


__global__ void os_menores(int *d_matrizes, int numMats, int jump) {
  int indexAtual = indexIni = threadIdx.x + blockIdx.x * blockDim.x;
  int tamMatriz = 9;

  if(index < numMats * tamMatriz) {

  }
  else {
    return;
  }

}

/* Imprime todas as matrizes de dimensão ExE contidas em matrizes*/
void leitura (int *matrizes) {
  int i, k;
  for (i = 0; i < numMatrizes * linhaElementos; i++) {
    for (k = 0; k < linhaElementos; k++) 
      printf("%d\t", *(matrizes++));
    printf("\n");
    if((i+1) % linhaElementos == 0)
      printf("********************\n");
  }
}


void menorMatriz(int *d_matrizes, int numMats) {
  int numBlocks = numMats / 30;
  int numMatResto;
  int numThreads = 27;




  os_menores<<<numBlocks, numThreads>>>(d_matrizes, numMats, numThreads * numBlocks);



  // if((numMatResto = numMats % 30) != 0) {
  //   // caso com chamada a mais

  // }



}


void encontraMenorMatriz(int* matrizes) {
  int tam = numMatrizes * E * sizeof(int);
  int *d_matrizes;

  // Alloc space for device copies of a, b, c
  hipMalloc((void **) &d_matrizes, tam);

  // Copy inputs to device
  hipMemcpy(d_matrizes, matrizes, tam, hipMemcpyHostToDevice);

  // encontra menor matriz
  menorMatriz(d_matrizes, numMatrizes);
  
  // Copy result back to host
  hipMemcpy(matrizes, d_matrizes, tam, hipMemcpyDeviceToHost);

  hipFree(d_matrizes);  
}


// int * get_min (int **mat, int ini, int fim) {
//  int *parteA, *parteB;
//  if (ini - fim > 1){ /* mais que 2 elementos*/
//      meio = (ini + fim)/2;
//      parteA = get_min(mat, ini, meio);
//      parteB = get_min(mat, meio + 1, fim);
//      return compara(parteA, parteB);
//  }else if (ini - fim == 1){ /* 2 elementos */
//      return compara(mat[ini], mat[fim]);
//  }
//  else {
//      return mat[ini];
//  }
// }

// void apaga (int **mat){
//  int i;
//  int linhas = (sizeof(mat)/sizeof(int *));
//  for (i = 0; i < linhas; ++i) {
//      free(mat[i]);
//  }
//  free(mat);
// }


/* Le o arquivo arq que contem matrizes no formato declarado
no enunciado e retorna um vetor com todas matrizes lidas*/
int* alocaMatrizesArquivo(FILE *arq){
  char asteriscos[10];
  int *matrizes, *matrizesAux;
  fscanf(arq, "%d", &numMatrizes);
  matrizes =  (int *) malloc(E * numMatrizes * sizeof(int));

  matrizesAux = matrizes;
  for(int i = 0; i < numMatrizes; i++) {
    fscanf(arq, "%s", asteriscos); //pula a linha de asteriscos
    for(int j = 0; j < E; j++)
      fscanf(arq, "%d", matrizesAux++);
  }
  leitura(matrizes);
  return matrizes;
}


int main (int argc, char* argv[]) {
  if(argc != 2) {
    printf("Argumento do programa: nome do arquivo\n");
  }
  else {  
    FILE *entrada;
    entrada =  fopen(argv[1], "r");
    if (entrada == NULL) {
      printf("Deu ruim pra abrir o arquivo\n");
      return EXIT_FAILURE;
    }
    int *matrizes = alocaMatrizesArquivo(entrada);

    fclose(entrada);
    // leitura(get_min(mat, 0, qtde));
    free(matrizes);
    return EXIT_SUCCESS;
  }
}

